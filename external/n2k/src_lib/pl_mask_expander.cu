#include "hip/hip_runtime.h"
#include "../include/n2k/pl_kernels.hpp"
#include "../include/n2k/internals/internals.hpp"

#include <gputils/cuda_utils.hpp>
#include <gputils/string_utils.hpp>

using namespace std;
using namespace gputils;

namespace n2k {
#if 0
}  // editor auto-indent
#endif


// FIXME can be improved.
__device__ inline uint2 double_bits(uint x)
{
    uint2 ret{0U,0U};
    
    for (uint i = 0; i < 16; i++) {
	uint x_shifted = x >> i;
	uint bit_pair = 3U << (2*i);
	
	if (x_shifted & 1U)
	    ret.x |= bit_pair;
	if (x_shifted & 0x10000U)
	    ret.y |= bit_pair;
    }

    return ret;
}


// Kernel args:
//   F = (number of output freqs)
//   M = (number of input times) / 64
//   N = (number of stations) * 2
//   pl_out = uint array of shape (2*M, F, N)
//   pl_in = uint array of shape (M, (F+3)/4, N)

__global__ void pl_mask_expand_kernel(uint *pl_out, const uint *pl_in, int F, int M, int N)
{
    const int Fin = (F+3) >> 2;
    
    // Parallelization: x <-> n, y <-> f, z <-> m
    int n = (blockIdx.x * blockDim.x) + threadIdx.x;
    int f = (blockIdx.y * blockDim.y) + threadIdx.y; 
    int m = (blockIdx.z * blockDim.z) + threadIdx.z;
    
    bool valid = (f < Fin) && (m < M) && (n < N);
    int nf_out = valid ? min(F-4*f,4) : 0;

    // Ensure array accesses are within bounds.
    f = (f < Fin) ? f : (Fin-1);
    m = (m < M) ? m : (M-1);
    n = (n < N) ? n : (N-1);
    
    // pl_in = uint array of shape (M, Fin, N)
    // After these shifts, 'pl_in' points to a scalar.
    
    pl_in += long(m) * long(Fin*N);
    pl_in += (f*N + n);

    // pl_out = uint array of shape (2*M, F, N)
    // After these shifts, 'pl_out' points to an array of shape (nf_out, 2) with strides (N, 1).

    int mout = 2*m + (n & 1);
    int nout = n & ~1;
    pl_out += long(mout) * long(F*N);
    pl_out += (4*f*N + nout);

    // Read input mask.
    uint x = *pl_in;
    uint2 y = double_bits(x);
	
    // Write (expanded) output mask.
    for (int i = 0; i < nf_out; i++)
	*((uint2 *) (pl_out + i*N)) = y;
}


void launch_pl_mask_expander(ulong *pl_out, const ulong *pl_in, long Tout, long Fout, long S, hipStream_t stream)
{
    // FIXME asserts -> exceptions.
    assert(pl_out != nullptr);
    assert(pl_in != nullptr);
    assert(Tout > 0);
    assert(Fout > 0);
    assert(S > 0);

    assert((Tout % 128) == 0);
    assert((S % 16) == 0);

    // FIXME check for 32-bit overflows.
    
    long M = Tout / 128;
    long N = S * 2;

    dim3 nblocks, nthreads;
    gputils::assign_kernel_dims(nblocks, nthreads, N, Fout, M);  // x <-> n, y <-> f, z <-> m

    pl_mask_expand_kernel <<< nblocks, nthreads, 0, stream >>>
	((uint *) pl_out, (const uint *) pl_in, Fout, M, N);

    CUDA_PEEK("pl_mask_expand_kernel");
}


void launch_pl_mask_expander(Array<ulong> &pl_out, const Array<ulong> &pl_in, hipStream_t stream)
{
    check_array(pl_out, "launch_pl_mask_expander", "pl_out", 3, true);  // contiguous=true
    check_array(pl_in, "launch_pl_mask_expander", "pl_in", 3, true);    // contiguous=true

    long Tout = 64 * pl_out.shape[0];
    long Fout = pl_out.shape[1];
    long S = pl_out.shape[2];

    if (!pl_in.shape_equals({Tout/128, (Fout+3)/4, S})) {
	stringstream ss;
	ss << "launch_pl_mask_expander: pl_out.shape=" << pl_out.shape_str()
	   << " and pl_in.shape=" << pl_in.shape_str()
	   << " are inconsistent (expected pl_in.shape=("
	   << (Tout/128) << "," << ((Fout+3)/4) << "," << S 
	   << "))";
	throw runtime_error(ss.str());
    }

    launch_pl_mask_expander(pl_out.data, pl_in.data, Tout, Fout, S, stream);
}


}  // namespace n2k
